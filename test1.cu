
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16     // block size

#define v3_v3_dot(a, b) (a.x * b.x + a.y * b.y + a.z * b.z)


__global__ void TestKernel( uchar4* dst,float3* normal_map, float3 cam_vec, unsigned int imgWidth, unsigned int imgHeight )
{
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int bw = blockDim.x;
    unsigned int bh = blockDim.y;
    // Non-normalized U, V coordinates of input texture for current thread.
    unsigned int u = ( bw * blockIdx.x ) + tx;
    unsigned int v = ( bh * blockIdx.y ) + ty;
    unsigned int index = ( v * imgWidth ) + u;



    float G = v3_v3_dot(normal_map[index], cam_vec);

    int g = G*127+127;

    dst[index] = make_uchar4(g, g, g, 255);

}



extern "C" void first_test(uchar4* g_dstBuffer, float3* normal_map,float3 cam_vec, int width, int height) {

	size_t blocksW = (size_t)ceilf( width / (float)BLOCK_SIZE );
	size_t blocksH = (size_t)ceilf( height / (float)BLOCK_SIZE );
	dim3 gridDim( blocksW, blocksH, 1 );
	dim3 blockDim( BLOCK_SIZE, BLOCK_SIZE, 1 );
	TestKernel<<< gridDim, blockDim >>>( g_dstBuffer, normal_map, cam_vec, width, height );


}